#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "structs.h"
#include "data.h"

void print_rawunits(rawunits_t *units) {
    size_t num_cols = units->covar_cnt + 1;

    for (size_t i = 0; i < units->unit_cnt; i++) {
        printf("%zu: [", i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", get_rawunit_column_val(units, i, j), j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_strata(strata *strata) {
    size_t num_cols = strata->data_colcnt;

    for (size_t i = 0; i < strata->stratum_cnt; i++) {
        printf("%zu: [", i);
        float *data_cols = get_stratum_start(strata, i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", data_cols[j], j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_stratamaps(covarstratamap_t *covar_strata_map) {
    stratamappings_t *strata_maps = covar_strata_map->strata_maps;
    strataindices_t *strata_ixs = covar_strata_map->strata_ixs;

    printf("Entries: %zu, allocated: %zu\n",
        strata_maps->entries_cnt,
        strata_maps->allocated_cnt);

    for (size_t i = 0; i < strata_maps->entries_cnt; i++) {
        printf("Entry: %zu\n", i);
        stratamapping_t *cur_mapping = &strata_maps->entries[i];
        printf("\tValue: %.2f, start: %zu, entries: %zu\n",
            cur_mapping->value,
            cur_mapping->start_index,
            cur_mapping->entries_cnt);
        printf("\tStratum indices:\n");

        for (size_t j = 0; j < cur_mapping->entries_cnt; j++) {
            printf("\t\t%zu\n",
                strata_ixs->indices[cur_mapping->start_index + j]);
        }
    }
}

int main(int argc, char **argv) {
    srand(time(NULL));

    size_t num_units = 20, num_covars = 3, num_groups = 2;

    rawunits_t *units = get_randomunits(num_units, num_covars, num_groups);
    sort_rawunits(units);

    printf("Raw units\n");
    print_rawunits(units);
    printf("\n");

    strata_t *strata = strata_from_sorted_rawunits(units);
    printf("Strata\n");
    print_strata(strata);
    printf("\n");

    covarstratamap_t *covar_strata_map = build_covarstratamap(strata);

    printf("Covar maps indices:\n");
    for (size_t i = 0; i < strata->covar_cnt; i++)
        printf("\tCovar: %zu, Start index: %zu\n", i, covar_strata_map->covar_map_indices[i]);
    printf("\n");

    printf("Strata maps\n");
    print_stratamaps(covar_strata_map);
    printf("\n");

    free_covarstratamap(covar_strata_map);

    free(strata);
    free(units);

    return 0;
}