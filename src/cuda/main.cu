#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "structs.h"
#include "data.h"

void print_rawunits(rawunits_t *units) {
    size_t num_cols = units->covar_cnt + 1;

    for (int i = 0; i < units->unit_cnt; i++) {
        printf("%d: [", i);
        for (int j = 0; j < num_cols; j++) {
            printf("%.2f%s", units->data[i * num_cols + j], j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

int main(int argc, char **argv) {
    srand(time(NULL));

    size_t num_units = 20, num_covars = 3, num_groups = 2;

    rawunits_t *units = get_randomunits(num_units, num_covars, num_groups);
    print_rawunits(units);

    sort_rawunits(units);

    printf("\nAfter sort\n");
    print_rawunits(units);

    free(units);

    return 0;
}