#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "structs.h"
#include "data.h"

void print_rawunits(rawunits_t *units) {
    size_t num_cols = units->covar_cnt + 1;

    for (size_t i = 0; i < units->unit_cnt; i++) {
        printf("%zu: [", i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", get_rawunit_column_val(units, i, j), j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_strata(strata *strata) {
    size_t num_cols = strata->data_colcnt;

    for (size_t i = 0; i < strata->stratum_cnt; i++) {
        printf("%zu: [", i);
        float *data_cols = get_stratum_start(strata, i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", data_cols[j], j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

int main(int argc, char **argv) {
    srand(time(NULL));

    size_t num_units = 20, num_covars = 3, num_groups = 2;

    rawunits_t *units = get_randomunits(num_units, num_covars, num_groups);
    sort_rawunits(units);

    printf("Raw units\n");
    print_rawunits(units);
    printf("\n");

    strata_t *strata = strata_from_sorted_rawunits(units);
    printf("Strata\n");
    print_strata(strata);

    free(strata);
    free(units);

    return 0;
}