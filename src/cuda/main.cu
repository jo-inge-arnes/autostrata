#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "structs.h"
#include "data.h"

void print_rawunits(rawunits_t *units) {
    size_t num_cols = units->covar_cnt + 1;

    for (size_t i = 0; i < units->unit_cnt; i++) {
        printf("%zu: [", i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", get_rawunit_column_val(units, i, j), j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_strata(strata *strata) {
    size_t num_cols = strata->data_colcnt;

    for (size_t i = 0; i < strata->stratum_cnt; i++) {
        printf("%zu: [", i);
        float *data_cols = get_stratum_start(strata, i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", data_cols[j], j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_stratamaps(covarstratamap_t *covar_strata_map) {
    stratamappings_t *strata_maps = covar_strata_map->strata_maps;
    strataindices_t *strata_ixs = covar_strata_map->strata_ixs;

    printf("Entries: %zu, allocated: %zu\n",
        strata_maps->entries_cnt,
        strata_maps->allocated_cnt);

    for (size_t i = 0; i < strata_maps->entries_cnt; i++) {
        printf("Entry: %zu\n", i);
        stratamapping_t *cur_mapping = &strata_maps->entries[i];
        printf("\tValue: %.2f, start: %zu, entries: %zu\n",
            cur_mapping->value,
            cur_mapping->start_index,
            cur_mapping->entries_cnt);
        printf("\tStratum indices:\n");

        for (size_t j = 0; j < cur_mapping->entries_cnt; j++) {
            printf("\t\t%zu\n",
                strata_ixs->indices[cur_mapping->start_index + j]);
        }
    }
}

int main(int argc, char **argv) {
    srand(time(NULL));

    size_t num_units = 20, num_covars = 3, num_groups = 2;

    rawunits_t *units = get_randomunits(num_units, num_covars, num_groups);
    sort_rawunits(units);

    printf("Raw units\n");
    print_rawunits(units);
    printf("\n");

    strata_t *strata = strata_from_sorted_rawunits(units);
    printf("Strata\n");
    print_strata(strata);
    printf("\n");

    covarstratamap_t *covar_strata_map = build_covarstratamap(strata);

    printf("Covar maps indices:\n");
    for (size_t i = 0; i < strata->covar_cnt; i++)
        printf("\tCovar: %zu, Start index: %zu\n", i, covar_strata_map->covar_map_indices[i]);
    printf("\n");

    printf("Strata maps\n");
    print_stratamaps(covar_strata_map);
    printf("\n");

    // Find neighbour strata
    for (size_t covar_index = 0; covar_index < strata->covar_cnt; covar_index++) {
        size_t num_vals;
        if (covar_index < strata->covar_cnt - 1) {
            num_vals = covar_strata_map->covar_map_indices[covar_index + 1] -
                covar_strata_map->covar_map_indices[covar_index];
        } else {
            num_vals = covar_strata_map->strata_maps->entries_cnt -
                covar_strata_map->covar_map_indices[covar_index];
        }

        for (size_t value_index = 1; value_index < num_vals; value_index++) {

            if (value_index > 0) {
                size_t covar_offset =
                    covar_strata_map->covar_map_indices[covar_index];
                stratamapping_t *cur_mapping =
                    &covar_strata_map->strata_maps->entries[covar_offset + value_index];

                stratamapping_t *prev_mapping =
                    &covar_strata_map->strata_maps->entries[covar_offset + value_index - 1];

                size_t upper_stratum_index, lower_stratum_index;
                for (size_t i = 0; i < cur_mapping->entries_cnt; i++) {
                    upper_stratum_index =
                        covar_strata_map->strata_ixs->indices[cur_mapping->start_index + i];

                    for (size_t j = 0; j < prev_mapping->entries_cnt; j++) {
                        lower_stratum_index =
                            covar_strata_map->strata_ixs->indices[prev_mapping->start_index + j];

                        if (are_equal_except(strata, upper_stratum_index, lower_stratum_index, covar_index)) {
                            printf("For covariate_%zu = %.2f, strata %zu and %zu are neighbours\n",
                                covar_index,
                                cur_mapping->value,
                                upper_stratum_index,
                                lower_stratum_index
                                );
                        }
                    }
                }
            } else {
                fprintf(stderr,
                    "No neighbour stratum possible on the lower side of the lowest edge "
                    "for the given covariate.\n");
            }
        }
    }

    free_covarstratamap(covar_strata_map);

    free(strata);
    free(units);

    return 0;
}