#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "structs.h"
#include "data.h"

void print_rawunits(rawunits_t *units) {
    size_t num_cols = units->covar_cnt + 1;

    for (size_t i = 0; i < units->unit_cnt; i++) {
        printf("%zu: [", i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", get_rawunit_column_val(units, i, j), j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_strata(strata *strata) {
    size_t num_cols = strata->data_colcnt;

    for (size_t i = 0; i < strata->stratum_cnt; i++) {
        printf("%zu: [", i);
        float *data_cols = get_stratum_start(strata, i);
        for (size_t j = 0; j < num_cols; j++) {
            printf("%.2f%s", data_cols[j], j < num_cols - 1 ? ", ": "");
        }
        printf("]\n");
    }
}

void print_stratamaps(stratamappings_t *strata_maps, strataindices_t *strata_ixs, strata_t *strata) {
    printf("Entries: %zu, allocated: %zu\n",
        strata_maps->entries_cnt,
        strata_maps->allocated_cnt);

    for (size_t i = 0; i < strata_maps->entries_cnt; i++) {
        printf("Entry: %zu\n", i);
        stratamapping_t *cur_mapping = &strata_maps->entries[i];
        printf("\tValue: %.2f, start: %zu, entries: %zu\n",
            cur_mapping->value,
            cur_mapping->start_index,
            cur_mapping->entries_cnt);
        printf("\tStratum indices:\n");

        for (size_t j = 0; j < cur_mapping->entries_cnt; j++) {
            printf("\t\t%zu\n",
                strata_ixs->indices[cur_mapping->start_index + j]);
        }
    }
}

int main(int argc, char **argv) {
    srand(time(NULL));

    size_t num_units = 20, num_covars = 3, num_groups = 2;

    rawunits_t *units = get_randomunits(num_units, num_covars, num_groups);
    sort_rawunits(units);

    printf("Raw units\n");
    print_rawunits(units);
    printf("\n");

    strata_t *strata = strata_from_sorted_rawunits(units);
    printf("Strata\n");
    print_strata(strata);
    printf("\n");

    stratamappings_t *strata_maps = allocate_stratamappings(strata->stratum_cnt * strata->covar_cnt);
    strataindices_t *strata_ixs = allocate_strataindices(strata->stratum_cnt * strata->covar_cnt);

    float cur_val, prev_val;
    float *stratum_vals;
    stratamapping_t *cur_mapping;

    size_t *covar_map_indices =
        (size_t*)malloc(sizeof(size_t) * strata->covar_cnt);

    for (size_t covar_index = 0; covar_index < strata->covar_cnt; covar_index++) {
        covar_map_indices[covar_index] = strata_maps->entries_cnt;

        valueindex_t *sorted = index_sort(strata, covar_index);
        stratum_vals = get_stratum_start(strata, sorted[0].index);
        cur_val = stratum_vals[covar_index];
        cur_mapping = &strata_maps->entries[strata_maps->entries_cnt];
        strata_maps->entries_cnt++;
        cur_mapping->value = cur_val;
        cur_mapping->start_index = strata_ixs->indices_cnt;
        cur_mapping->entries_cnt++;
        strata_ixs->indices[strata_ixs->indices_cnt] = sorted[0].index;
        strata_ixs->indices_cnt++;
        prev_val = cur_val;

        for (size_t i = 1; i < strata->stratum_cnt; i++) {
            stratum_vals = get_stratum_start(strata, sorted[i].index);
            cur_val = stratum_vals[covar_index];

            if (cur_val != prev_val) {
                cur_mapping = &strata_maps->entries[strata_maps->entries_cnt];
                strata_maps->entries_cnt++;
                cur_mapping->value = cur_val;
                cur_mapping->start_index = strata_ixs->indices_cnt;
            }

            cur_mapping->entries_cnt++;
            strata_ixs->indices[strata_ixs->indices_cnt] = sorted[i].index;
            strata_ixs->indices_cnt++;

            prev_val = cur_val;
        }

        free(sorted);
    }

    strata_maps = shrink_stratamappings(strata_maps);
    strata_ixs = shrink_strataindices(strata_ixs);

    printf("Covar maps indices:\n");
    for (size_t i = 0; i < strata->covar_cnt; i++)
        printf("\tCovar: %zu, Start index: %zu\n", i, covar_map_indices[i]);
    printf("\n");

    printf("Strata maps\n");
    print_stratamaps(strata_maps, strata_ixs, strata);
    printf("\n");

    free(strata_ixs);
    free(strata_maps);
    free(covar_map_indices);

    free(strata);
    free(units);

    return 0;
}